
#include <hip/hip_runtime.h>

// Permite calcular o m�ximo divisor comum entre dois n�meros inteiros.
extern "C" __device__ int GreatCommonDivisorInt(int x, int y){
	int innerx = x, innery = y, aux = 0;
	while (innery != 1)
	{
		aux = innery;
		innery = innerx % innery;
		innerx = aux;
	}

	return innerx;
}

// Permite calcular o m�ximo divisor comum entre dois n�meros inteiros sem sinal.
extern "C" __device__ unsigned int GreatCommonDivisorUnsignedInt(unsigned int x, unsigned int y){
	int innerx = x, innery = y, aux = 0;
	while (innery != 1)
	{
		aux = innery;
		innery = innerx % innery;
		innerx = aux;
	}

	return innerx;
}

// Permite calcular o m�ximo divisor comum entre dois n�meros longos.
extern "C" __device__ long GreatCommonDivisorLong(long x, long y){
	int innerx = x, innery = y, aux = 0;
	while (innery != 1)
	{
		aux = innery;
		innery = innerx % innery;
		innerx = aux;
	}

	return innerx;
}

// Permite calcular o m�ximo divisor comum entre dois n�meros longos sem sinal.
extern "C" __device__ unsigned long GreatCommonDivisorUnsignedLong(unsigned long x, unsigned long y){
	int innerx = x, innery = y, aux = 0;
	while (innery != 1)
	{
		aux = innery;
		innery = innerx % innery;
		innerx = aux;
	}

	return innerx;
}