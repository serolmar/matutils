#include "hip/hip_runtime.h"
#include "SequentialDeviceFuncs.h"

/// <summary>
/// Adi��o de um vector de inteiros a outro, sendo o resultado estabelecido no par�metro a.
/// </summary>
/// <param name="a">O primeiro vector a ser adicionado.</param>
/// <param name="b">O segundo vector a ser adicionado.</param>
extern "C" __global__  void AddInt(int* a, int *b)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	a[x] += b[x];
}

/// <summary>
/// Adi��o de um vector de inteiros sem sinal a outro, sendo o resultado estabelecido no par�metro a.
/// </summary>
/// <param name="a">O primeiro vector a ser adicionado.</param>
/// <param name="b">O segundo vector a ser adicionado.</param>
extern "C" __global__  void AddUnsignedInt(unsigned int* a, unsigned int *b)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	a[x] += b[x];
}

/// <summary>
/// Adi��o de um vector de longos a outro, sendo o resultado estabelecido no par�metro a.
/// </summary>
/// <param name="a">O primeiro vector a ser adicionado.</param>
/// <param name="b">O segundo vector a ser adicionado.</param>
extern "C" __global__  void AddLong(long* a, long *b)
{
	int x = blockIdx.x;
	a[x] += b[x];
}

/// <summary>
/// Adi��o de um vector de longos sem sinal a outro, sendo o resultado estabelecido no par�metro a.
/// </summary>
/// <param name="a">O primeiro vector a ser adicionado.</param>
/// <param name="b">O segundo vector a ser adicionado.</param>
extern "C" __global__  void AddUnsignedLong(unsigned long * a, unsigned long *b)
{
	int x = blockIdx.x;
	a[x] += b[x];
}

/// <summary>
/// Adi��o de um vector de v�rgula flutuante de precis�o simples a outro, sendo o resultado 
/// estabelecido no par�metro a.
/// </summary>
/// <param name="a">O primeiro vector a ser adicionado.</param>
/// <param name="b">O segundo vector a ser adicionado.</param>
extern "C" __global__ void AddFloat(float *a, float *b){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	a[x] += b[x];
}

/// <summary>
/// Adi��o de um vector de v�rgula flutuante de precis�o simples a outro, sendo o resultado 
/// estabelecido no par�metro a.
/// </summary>
/// <param name="a">O primeiro vector a ser adicionado.</param>
/// <param name="b">O segundo vector a ser adicionado.</param>
extern "C" __global__  void AddDouble(double* a, double *b)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	a[x] += b[x];
}

/// <summary>
/// Permite adicionar vectores de frac��es, tendo em conta que cada vector cont�m inteiros
/// relativos ao numerador e ao denominador.
/// </summary>
/// <param name="a">O primeiro vector a ser adicionado.</param>
/// <param name="b">O segundo vector a ser adicionado.</param>
extern "C" __global__ void AddIntegerFraction(int* a, int * b){
	int gcd;
	int nextx;
	int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
	nextx = x + 1;

	// Efectua a multiplica��o simples das frac��es.
	a[x] *= b[nextx];
	a[x] += b[x] * a[x];
	a[nextx] *= b[nextx];

	// Aplica o m�ximo divisor comum a ambos os itens.
	gcd = GreatCommonDivisorInt(a[x], a[nextx]);
	a[x] /= gcd;
	a[nextx] /= gcd;
}

/// <summary>
/// Permite adicionar vectores de frac��es, tendo em conta que cada vector cont�m 2 * length inteiros sem sinal
/// relativos ao numerador e ao denominador.
/// </summary>
/// <param name="a">O primeiro vector a ser adicionado.</param>
/// <param name="b">O segundo vector a ser adicionado.</param>
extern "C" __global__ void AddUnsignedIntegerFraction( unsigned int* a, unsigned int* b){
	unsigned int gcd;
	int nextx;
	int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
	nextx = x + 1;

	// Efectua a multiplica��o simples das frac��es.
	a[x] *= b[nextx];
	a[x] += b[x] * a[x];
	a[nextx] *= b[nextx];

	// Aplica o m�ximo divisor comum a ambos os itens.
	gcd = GreatCommonDivisorUnsignedInt(a[x], a[nextx]);
	a[x] /= gcd;
	a[nextx] /= gcd;
}

/// <summary>
/// Permite adicionar vectores de frac��es, tendo em conta que cada vector cont�m 2 * length longos
/// relativos ao numerador e ao denominador.
/// </summary>
/// <param name="a"></param>
/// <param name="b"></param>
extern "C" __global__ void AddLongFraction(long* a, long* b){
	long gcd;
	long nextx;
	int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
	nextx = x + 1;

	// Efectua a multiplica��o simples das frac��es.
	a[x] *= b[nextx];
	a[x] += b[x] * a[x];
	a[nextx] *= b[nextx];

	// Aplica o m�ximo divisor comum a ambos os itens.
	gcd = GreatCommonDivisorLong(a[x], a[nextx]);
	a[x] /= gcd;
	a[nextx] /= gcd;
}

/// <summary>
/// Permite adicionar vectores de frac��es, tendo em conta que cada vector cont�m 2 * length longos sem sinal
/// relativos ao numerador e ao denominador.
/// </summary>
/// <param name="a"></param>
/// <param name="b"></param>
extern "C" __global__ void AddUnsignedLongFraction(unsigned long* a, unsigned long* b){
	unsigned long gcd;
	unsigned long nextx;
	int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
	nextx = x + 1;

	// Efectua a multiplica��o simples das frac��es.
	a[x] *= b[nextx];
	a[x] += b[x] * a[x];
	a[nextx] *= b[nextx];

	// Aplica o m�ximo divisor comum a ambos os itens.
	gcd = GreatCommonDivisorLong(a[x], a[nextx]);
	a[x] /= gcd;
	a[nextx] /= gcd;
}

/// <summary>
/// Permite calcular o produto escalar de tantas sec��es de dois vectores quantos os blocos de chamada.
/// </summary>
/// <remarks>
/// O processo de determina��o do produto escalar baseia-se na redu��o por blocos e o tamanho dos vectores
/// ter� de ser uma pot�ncia de dois.
/// Ver http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
/// </remarks>
/// <param name="a"></param>
/// <param name="b"></param>
/// <param name="c"></param>
extern "C" __global__ void InnerProdIntegerVectorRed(int* a, int* b, int* c){
	// A mem�ria est�tica ter� de ser reservada no c�digo de anfitri�o de modo a poder ser utilizada
	extern __shared__ int* innera;
	extern __shared__ int* innerb;

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	innera[tid] = a[tid];
	innerb[tid] = b[tid];
	__syncthreads();
}